#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>


#define PI   3.1415
#define L    1.0          // the length of the box
#define DIM  500          // the dimensions of the window
#define E    (L/DIM)      // epsilon
#define HBAR 1.054        // without the e-34

#define MAX(a,b) a > b ? a : b
#define MIN(a,b) a < b ? a : b
#define ERROR(msg) do { fprintf(stderr, "%s(%d): %s\n", __FILE__, __LINE__, msg); exit(0); } while(0);


typedef struct _wave_function {
    int n;             // the energy level of the wave function
    float energy;      // the energy of the particle at that level
    float probability; // the probability of getting this wave function (c^2_n)
} _wave_function;

typedef struct particle {
    float mass;            // the mass of the particle
    _wave_function **wave; // array of wave functions for the particle 
    int waveCount;
} particle;



/**
 * Create a new wave
 *
 * @param _wave_function  the wave function to create
 * @param int             the energy level of the particle
 * @param float           the mass of the particle
 * @param float           the probability of the wave function (c^2_n)
*/
void _create_wave(_wave_function *wave, int n, float mass, float p) {
    wave->n = n;
    wave->energy = HBAR*HBAR * PI*PI * n*n / (mass * L*L);
    wave->probability = p;
}

/**
 * Get the probability of finding a particle in one dimension
 *
 * @param particle      the particle to find the probability of
 * @param float         the position in which to find the probability at
 * @return              the probability of finding the particle at the point x
*/
float _probability_1d(_wave_function *w, float x) {
    int n   =  w->n;
    float s =  E/L - (1/(n*PI)) * cos(2*n*PI*x/L) * sin(n*PI / DIM);
    return s;
}


/**
 * Given numbers, the function generates the next set of probabilities
 * based on these numbers.
 *
 * @param int           the numbers to generate the probabilities from
 * @param int           the length of the numbers array (should be the same
                        for the probabilities array);
 * @param float         array of probabilities to write to
*/
void next_probabilities(int *numbers, int N, float *probabilities) {
    int sum = 0;
    int carry = 1;
    for(int i=N-1;i>=0;i--) {
        numbers[i] +=  carry;
        carry       =  numbers[i] / 10;
        numbers[i] %=  10;
        sum        +=  numbers[i];
    }
    for(int i=0;i<N;i++) {
        probabilities[i] = 1.0f*numbers[i] / sum;
    }
}

/**
 * Create a new particle
 * 
 * @param particle      the particle to create
 * @param int           the number of energy levels
 * @param float         array of the probability of every energy level
 * @param float         the mass of the particle
*/
void create_particle(particle *p, int N, float *probabilities, float mass) {
    p->mass = mass;
    p->waveCount = N;
    p->wave = (_wave_function **) malloc(N*sizeof(_wave_function));
    for(int i=0;i<N;i++) {
        p->wave[i] = (_wave_function *) malloc(sizeof(_wave_function));
        _create_wave(p->wave[i], i+1, mass, probabilities[i]);
    }
}

/**
 * Get the probability of finding a particle in two dimensions
 *
 * @param particle      the particle to find the probability of
 * @param float         the x-position in which to find the probability at
 * @param float         the y-position in which to find the probability at
 * @return              the probability of finding the particle at the point (x, y)
*/
float probability(particle *p, float x, float y) {
    if(x > 0 && x < L && y > 0 && y < L) {
        float probability = 0;
        for(int i=0;i<p->waveCount;i++)
            probability += (1.0/(L*L) * _probability_1d(p->wave[i], x) *
                           _probability_1d(p->wave[i], y)) * p->wave[i]->probability;
        return probability;
    }
    return 0;
}



int main(int argc, const char *argv[]) { 

    particle p;
    int N = 10;
    float *pro = (float*) malloc(N*sizeof(float));
    for(int i=0;i<N;i++) pro[i] = 0.1f;
    create_particle(&p, N, pro, 2.5);

    printf("Testing on a particle\n");
    printf("The particle has mass %f and has %d energy levels\n",
           p.mass, p.waveCount);
    for(int i=0;i<p.waveCount;i++)
        printf("\tWave %d has %f energy and %.3f probability\n", i+1, p.wave[i]->energy,
               p.wave[i]->probability);

    float all_proba = 0;
    float max_proba = 0, max_x, max_y;
    for(float x=0;x<=L;x+=E) {
        for(float y=0;y<=L;y+=E) {
            float pn = probability(&p, x, y);
            if(pn > max_proba) {
                max_proba = pn;
                max_x = x;
                max_y = y;
            }
            all_proba += probability(&p, x, y);
        }
    }
    printf("The probability inside the box is %f\n", all_proba);

    float x = 0.5,
          y = 0.5;
    printf("The maximum probability is %f%% at (%.3f, %.3f)\n", max_proba*100, max_x, max_y);
    printf("The probability of finding the particle in (%f, %f) is %.5f%% or %.3f%% of the maximum\n",
           x, y, probability(&p, x, y)*100, probability(&p, x, y)*100/max_proba);

    printf("Testing on generating new probabilities\n");
    int *numbers = (int*) malloc(N*sizeof(int));
    float *proba = (float*) malloc(N*sizeof(float));
    for(int i=0;i<N;i++) numbers[i] = 0;
    numbers[0] = 1;
    for(int i=0;i<10;i++) {
        next_probabilities(numbers, N, proba);
        for(int i=0;i<N;i++)
            printf("%.3f\t", proba[i]);
        printf("\n");
    }
    return 0;
}
