#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <curses.h>

#define GL_GLEXT_PROTOTYPES
#include <GL/glut.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>


#define PI   3.141592653589793
#define L    1.0            // the length of the box
#define DIM  512            // the dimensions of the window (should be 2^n)
#define E    (L/DIM)        // epsilon
#define HBAR 1.054          // without the e-34


#define MAX(a,b) a > b ? a : b
#define MIN(a,b) a < b ? a : b


/* The struct that describes a particle */
typedef struct particle {
    float mass;            // mass of the particle
    int energy_levels;     // the number of energy levels
    float *probabilities;  // the probabilitis of every energy level
} particle;


/** global variables used in the code */
hipGraphicsResource *resource;
GLuint   buffer;
float    INCREASE_TIME = 0.01;           // number to increase the time by
particle p;                              // the particle
float    t             = INCREASE_TIME;  // starting time
int      frames        = 0;              // number of frames
float    total_time    = 0.0f;           // total time
int      PAUSE         = 0;              // whether the animation is paused

/** ncurses variables */
WINDOW* window;
int ncols, nrows;


/**
 * Function headers
*/
float max(float*, int);
void  next_probabilities(float, int, float*);
void  create_particle(particle*, int, float);
float probability(particle*, float, float);
float max_probability(particle*);
void  initGL(int*, char**);
void  display();
void  key(unsigned char, int, int);
void  free_resources();
void  createVBO(GLuint*, hipGraphicsResource**, unsigned int);
void  runCuda(hipGraphicsResource**);
void  launch_kernel(uchar4);
void  runCuda(hipGraphicsResource **resource);
void  run(int, char**);
void  usage(char*);
/** For ncurses */
void  clear_row(int);
void  cmd_display();
void  init_curses();


/**
 * CUDA kernel that finds the maximum in buckets of numbers
 * and fills it in the partialMax array using the reduction method
 *
 * @param float*            the numbers to find the maximum of
 * @param int               the length of the array (param 1)
 * @param float*            list of buckets containing the maximums
*/
__global__
void cuda_max(float *numbers, int N, float *partialMax) {
    extern __shared__ float cache[];
    int cacheIndex = threadIdx.x;
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    float m = 0;
    while(tid < N) {
        m = MAX(m, numbers[tid]);
        tid += blockDim.x * gridDim.x;
    }

    cache[cacheIndex] = m;
    __syncthreads();

    int i = blockDim.x/2;
    while(i != 0) {
        if(cacheIndex < i)
            cache[cacheIndex] = MAX(cache[cacheIndex], cache[cacheIndex + i]);
        __syncthreads();

        i /= 2;
    }

    if(cacheIndex == 0)
        partialMax[blockIdx.x] = cache[0];
}

/**
 * Function that finds the maximum of an array of numbers using the
 * cuda_max function
 *
 * @param float*            the array of numbers to find the maximum of
 * @param int               the length of the array
 * @return float            the maximum of the array
*/
float max(float *numbers, int N) {
    int tpb = 256; // threads per block
    int bpg = MIN(32, (N+tpb-1)/tpb); // blocks per grid

    float *partialMax = (float *) malloc(bpg*sizeof(float));
    float *devNumbers, *devPartialMax;

    hipMalloc((void**) &devNumbers, N*sizeof(float));
    hipMalloc((void**) &devPartialMax, bpg*sizeof(float));
    
    hipMemcpy(devNumbers, numbers, N*sizeof(float), hipMemcpyHostToDevice);
    cuda_max<<<tpb, bpg, bpg*sizeof(float)>>>(devNumbers, N, devPartialMax);
    hipMemcpy(partialMax, devPartialMax, bpg*sizeof(float), hipMemcpyDeviceToHost);

    float m = partialMax[0];
    for(int i=1;i<bpg;i++)
        m = MAX(m, partialMax[i]);

    hipFree(devNumbers);
    hipFree(devPartialMax);
    free(partialMax);
    free(numbers);
    return m;
}

/**
 * The set of probabilities can be expressed as a bunch of sine waves
 * For more information read section 4.4 (Next set of probabilities) on
 * page 9 in the paper
 *
 * @param float               the time
 * @param int                 the length of the array
 * @param float*              where to store the next set of probabilities
*/
void next_probabilities(float t, int N, float* probabilities) {
    float sum = 0;
    for(int i=0;i<N;i++) {
        probabilities[i] = abs(sin(pow(10, 1-i)*t));
        sum += probabilities[i];
    }
    // Normalize the probabilities
    for(int i=0;i<N;i++) probabilities[i] /= sum;
}

/**
 * A function that creates a new particle
 *
 * @param particle*           the particle to create
 * @param int                 the number of energy levels the particle can have
 * @param float               the mass of the particle
*/
void create_particle(particle *p, int N, float mass) {
    p->mass = mass;
    p->energy_levels = N;
    p->probabilities = (float*) malloc(N*sizeof(float));
    for(int i=0;i<N;i++)
        p->probabilities[i] = (i == 0); // 1 if i == 0, 0 otherwise
                                        // so sum(probabilities) = 1
}

/**
 * CUDA device function that finds the probability of finding the
 * particle in one dimension at the position x at the energy level n
 *
 * @param int                  the energy level of the particle
 * @param float                the position of the particle
 * @return float               the probability
*/
__device__
float cuda_probability_1d_device(int n, float x) {
    float s =  E/L - (1/(n*PI)) * cos(2*n*PI*x/L) * sin(n*PI / DIM);
    return s;
}

/**
 * CUDA device function that finds the probabiltiy of finding the 
 * particle at a fixed position given a set of probabilities, the
 * number of energy levels and the position
 *
 * @param float*                the probability of each energy level
 * @param int                   the number of energy levels
 * @param float                 the x-coordinate of the particle
 * @param float                 the y-coordinate of the particle
 * @return float                the probability
*/
__device__
float cuda_probability_2d_device(float *probability, int n, float x, float y) {
    float s = 0;
    for(int i=0;i<n;i++) {
        s += (1.0/(L*L) * cuda_probability_1d_device(i+1, x) *
                          cuda_probability_1d_device(i+1, y)) * probability[i];
    }
    return s;
}

/**
 * CUDA kernel that maps the coordinate array to a probability array
 *
 * @param float*                 the probability set of each energy level
 * @param int                    the number of energy levels
 * @param float*                 the array to map to
*/
__global__ 
void cuda_probability_to_map(float *probabilities, int n, float *map) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = gridDim.x * blockDim.x;

    while(i < DIM * DIM) {
        float x = 1.0*L*((int) (i%DIM))/DIM;
        float y = 1.0*L*((int) (i/DIM))/DIM;

        if(x > 0 && x < L && y > 0 && y < L)
            map[i] = cuda_probability_2d_device(probabilities, n, x, y);
        else
            map[i] = 0;
        i += offset;
    }
}

/**
 * CUDA kernel to find the probability of finding the particle at a certain
 * position
 *
 * @param float*                  the probability set of each energy level
 * @param int                     the number of energy levels
 * @param float                   the x-coordinate of the particle
 * @param float                   the y-coordinate of the particle
 * @param float*                  used to write the probability to
*/
__global__
void cuda_probability(float *p, int N, float x, float y, float *probability) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = blockDim.x * gridDim.x;

    if(i == 0)
        probability[0] = 0;
    __syncthreads();
    
    while(i < N) {
        if(x > 0 && x < L && y > 0 && y < L) {
            atomicAdd(&probability[0],  ( 1.0/(L*L) *
                                          cuda_probability_1d_device(i+1, x) *
                                          cuda_probability_1d_device(i+1, y)
                                        ) * p[i]);
        } else {
            probability[0] = 0;
        }
        i += offset;
    }
}

/**
 * A function that finds the probability of finding the particle at a certain position
 * @param particle* p               the particle
 * @param float                     the x-coordinate of the particle
 * @param float                     the y-coordinate of the particle
 * @return                          the probability
*/
float probability(particle *p, float x, float y) {
    float *devProbabilities, *devProbability, *probability;

    hipMalloc((void**) &devProbabilities, p->energy_levels*sizeof(float));
    hipMalloc((void**) &devProbability, sizeof(float));
    probability = (float*) malloc(sizeof(float));

    hipMemcpy(devProbabilities, p->probabilities, p->energy_levels*sizeof(float), hipMemcpyHostToDevice);
    cuda_probability<<<1, p->energy_levels>>>(devProbabilities, p->energy_levels, x, y, devProbability);
    hipMemcpy(probability, devProbability, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(devProbabilities);
    hipFree(devProbability);
    float *proba;
    proba = &probability[0];
    free(probability);
    return *proba;
}

/**
 * A function that finds the maximum probability in the space of finding the particle
 * This function is used to map the probabilities [0, max] |-> [0, 255]
 * 
 * @param particle*                  the particle
 * @return                           the highest probability
*/
float max_probability(particle *p) {
    float *devProbabilities, *devMap;
    float *map;

    map = (float*) malloc(DIM * DIM * sizeof(float));
    hipMalloc((void**) &devProbabilities, p->energy_levels * sizeof(float));
    hipMalloc((void**) &devMap, DIM * DIM * sizeof(float));
    
    hipMemcpy(devProbabilities, p->probabilities, p->energy_levels * sizeof(float), hipMemcpyHostToDevice);
    cuda_probability_to_map<<<32, 256>>>(devProbabilities, p->energy_levels, devMap);
    hipMemcpy(map, devMap, DIM * DIM * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(devProbabilities);
    hipFree(devMap);

    return max(map, DIM*DIM);

    /*
    // This algorithm failed for many waves
    // I had to fall back to the original bruteforce algorithm written above
    float probability = E/L, p_1d = 0.0;
    int n;
    for(int i=0;i<10;i++) {
        n = i+1;
        p_1d = E/L + 1.0/(n*PI) * sin(n*PI*E);
        probability += p->probabilities[i] * p_1d * p_1d;
    }

    return probability;
    */
}

/**
 * A function that finds the energy of the particle at a precise energy level
 *
 * @param float                     the mass of the particle
 * @param int                       the energy level
*/
__device__
float energy(float mass, int n) {
    return (HBAR * HBAR * PI * PI * n * n) / (mass * L * L);
}

/**
 * A function that finds the highest energy the particle can reach
 * This function is useful for color mapping
 *
 * @param float                     the mass of the particle
 * @return float                    the highest energy
*/
__device__
float highest_energy(float mass, int n) {
    return energy(mass, n);
}

__global__
void kernel(uchar4 *ptr, float *probabilities, int N, float max_proba) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = blockDim.x * gridDim.x;

    while(i < DIM*DIM) {
        float x = L*((int) (i%DIM))/DIM;
        float y = L*((int) (i/DIM))/DIM;
        float p = cuda_probability_2d_device(probabilities, N, x, y)/max_proba;
        float e = 0;
        for(int j=0;j<N;j++) e += probabilities[j]*energy(1.0, j+1);
        e /= highest_energy(1.0, N);

        ptr[i].x = 255*p*e;
        ptr[i].y = 20*p;
        ptr[i].z = 255*p*(1-e);

        i += offset;
    }
}


/////////////////////////// GUI PART /////////////////////////////////////
/**
 * Initialize the OpenGL environment
 *
 * @param int       length of next paramater
 * @param char      the parameters to the environment
*/
void initGL(int *argc, char **argv) {
    glutInit(argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(DIM, DIM);
    glutCreateWindow("Particle in a box simulation");
    glutDisplayFunc(display);

    glClearColor(0.0, 0.0, 0.0, 1.0);
    glDisable(GL_DEPTH_TEST);
}

/**
 * Manage key strokes
 *
 * @param unsigned char         the character pressed
 * @param int                   x-location of the pressing
 * @param int                   y-location of the pressing
*/
void key(unsigned char k, int x, int y) {
    switch(k) {
        case 27:
            free_resources();
            endwin();
            exit(0);
        case '.':
            INCREASE_TIME *= 1.05; break;
        case ',':
            INCREASE_TIME = MAX(0.01, INCREASE_TIME*0.95); break;
        case '0':
            t = 0.01f; break;
        case ' ':
            PAUSE = 1-PAUSE;
            if(PAUSE == 1) {
                glutPostRedisplay();
            } break;
        case 'm':
            t += INCREASE_TIME; glutPostRedisplay(); break;
        case 'n':
            t -= INCREASE_TIME; glutPostRedisplay(); break;
    }
}

/**
 * Free the OpenGL resources
*/
void free_resources() {
    hipGraphicsUnregisterResource(resource);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);
    glDeleteBuffers(1, &buffer);
}

/**
 * Function that creates the buffer and the resource for the environment
 *
 * @param GLuint                    the buffer used by OpenGL
 * @param hipGraphicsResource      the cuda resource to link to the buffer
*/
void createVBO(GLuint *buffer, hipGraphicsResource **resource,
               unsigned int flags) {
    glGenBuffers(1, buffer);
    glBindBuffer(GL_ARRAY_BUFFER, *buffer);

    unsigned int size = DIM * DIM * 4 * sizeof(float);
    glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    hipGraphicsGLRegisterBuffer(resource, *buffer, flags);
}

/**
 * Kernel launcher
 *
 * @param uchar4                the pixel array
*/
void launch_kernel(uchar4 *pos) {
    float *devProbabilities;
    int N = p.energy_levels;
    hipMalloc((void**) &devProbabilities, N*sizeof(float));
    hipMemcpy(devProbabilities, p.probabilities, N*sizeof(float), hipMemcpyHostToDevice);
    kernel<<<32, 256>>>(pos, devProbabilities, N, max_probability(&p));
    next_probabilities(t, N, (p.probabilities));
    hipFree(devProbabilities);
}

/**
 * Function that creates the resources for the kernel and launches it
 *
 * @param hipGraphicsResource  the cuda resource
*/
void runCuda(hipGraphicsResource **resource) {
    uchar4 *devPtr;
    size_t size;

    hipGraphicsMapResources(1, resource, 0);
    hipGraphicsResourceGetMappedPointer((void**) &devPtr, &size, *resource);
    launch_kernel(devPtr);
    hipGraphicsUnmapResources(1, resource, 0);
}


/**
 * The function that runs everything
 *
 * @param int       length of next paramater
 * @param char      the parameters to the environment
*/
void run(int argc, char **argv) {
    hipDeviceProp_t prop;
    int dev;
    memset(&prop, 0, sizeof(hipDeviceProp_t));
    prop.major = 1;
    prop.minor = 0;
    hipChooseDevice(&dev, &prop);
    cudaGLSetGLDevice(dev);

    initGL(&argc, argv);

    glutDisplayFunc(display);
    glutKeyboardFunc(key);
    createVBO(&buffer, &resource, cudaGraphicsMapFlagsWriteDiscard);
    runCuda(&resource);

    glutMainLoop();
}


/**
 * Prints out the help message of the program
 *
 * @param char      the name of the program
*/
void usage(char* program_name) {
    int y = 7+p.energy_levels/2+2;
    attron(COLOR_PAIR(2));
    move(y, 0);
    clear_row(y);
    printw("Keyboard Shortcuts");
    attroff(COLOR_PAIR(2));
    move(y+1, 0);
    attron(COLOR_PAIR(5));
    printw(".\t\tIncrease the time offset\n");
    printw(",\t\tDescrease the time offset\n");
    printw("n\t\tGo back one frame\n");
    printw("m\t\tGo forward one framed\n");
    printw("<space>\t\tToggle pausing\n");
    printw("0\t\tReset the animation\n");
    printw("<esc>\t\tQuit\n\n");
    attroff(COLOR_PAIR(5));
    attron(COLOR_PAIR(2));
    clear_row(y+9);
    printw("Usage");
    attroff(COLOR_PAIR(2));
    move(y+10, 0);
    attron(COLOR_PAIR(5));
    printw("%s [n]\nDESCRIPTION\n  n\n    The number of wave functions to simulate", program_name);
    attroff(COLOR_PAIR(5));
}

/**
 * Function that clears the rows
 */
void clear_row(int y) {
    move(y, 0);
    for(int i=0;i<ncols;i++) {
        move(y, i);
        delch();
        insch(' ');
    }
    move(y, 0);
}

/**
 * Create the command line interface
*/
void cmd_display() {
    // Draw the title
    attron(COLOR_PAIR(2));
    clear_row(0);
    printw("Simulating a two dimensional box in a square quantum box");

    // Draw the stats about the program
    attron(COLOR_PAIR(4));
    move(2, 0); printw("Simulating with %d wave functions", p.energy_levels);
    attroff(COLOR_PAIR(4));
    attron(COLOR_PAIR(2));
    clear_row(3);
    move(3, 0); printw("Time");
    move(3, 1*ncols/4); printw("Frame");
    move(3, 2*ncols/4); printw("Time per frame");
    move(3, 3*ncols/4); printw("Increment");
    attroff(COLOR_PAIR(2));
    attron(COLOR_PAIR(5));
    move(4, 0); printw("%.2f", t);
    move(4, 1*ncols/4); printw("%d", frames);
    move(4, 2*ncols/4); printw("%.2fms", total_time/frames);
    move(4, 3*ncols/4); printw("%.5f", INCREASE_TIME);
    attroff(COLOR_PAIR(5));

    // Write out the probabilities
    attron(COLOR_PAIR(2));
    move(6, 0);
    clear_row(6);
    for(int i=0;i<4;i++) {
        move(6, i*ncols/4);
        printw(i%2 == 0 ? "Wave number" : "Probability");
    }
    attroff(COLOR_PAIR(2));
    move(6, ncols/2-ncols/16);
    delch();
    insch(' ');

    attron(COLOR_PAIR(5));
    for(int i=0;i<p.energy_levels;i++) {
        move(7+i/2, i%2 * ncols/2);
        printw("%d", i+1);
        move(7+i/2, i%2 * ncols/2 + ncols/4);
        printw("%.2f%%", 100*p.probabilities[i]);
    }
    attroff(COLOR_PAIR(5));

    move(0, ncols);
    refresh();
}

/**
 * The display function that runs on every iteration
*/
void display() {
    hipEvent_t start, stop;
    hipEventCreate(&stop);
    hipEventCreate(&start);
    hipEventRecord(start, 0);
    
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, buffer);
    glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, DIM*DIM*4, NULL, GL_DYNAMIC_DRAW_ARB);
    hipGraphicsGLRegisterBuffer(&resource, buffer, cudaGraphicsMapFlagsNone);
    glClear(GL_COLOR_BUFFER_BIT);
    runCuda(&resource);
    glDrawPixels(DIM, DIM, GL_RGBA, GL_UNSIGNED_BYTE, 0);

    glutSwapBuffers();

    t += (1-PAUSE)*INCREASE_TIME;
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float diff;
    hipEventElapsedTime(&diff, start, stop);

    glutPostRedisplay();

    if(!PAUSE) {
        total_time += diff;
        frames++;
    }

    hipEventDestroy(start);
    hipEventDestroy(stop);

    cmd_display();
}

/**
 * Function that initializes the curses library
 * Sets up colors and fills the background
*/
void init_curses() {
    /** Initialize the ncurses variables */
    window = initscr();
    start_color();
    cbreak();
    noecho();
    getmaxyx(window, nrows, ncols);

    /** Initialize the colors that will be used in ncurses */
    init_pair(1, COLOR_BLACK, COLOR_BLACK); // background
    init_pair(2, COLOR_WHITE, COLOR_BLUE);  // titles
    init_pair(3, COLOR_RED, COLOR_BLACK);   // errors
    init_pair(4, COLOR_GREEN, COLOR_BLACK); // messages
    init_pair(5, COLOR_WHITE, COLOR_BLACK); // text

    clear();
    // Fill the background
    attron(COLOR_PAIR(1));
    for(int x=0;x<nrows;x++) {
        for(int y=0;y<ncols;y++) {
            move(x, y);
            delch();
            insch(' ');
        }
    }
    attroff(COLOR_PAIR(1));
    refresh();
}

///////////////////////////// MAIN FUNCTION //////////////////////////////////
int main(int argc, char *argv[]) { 
    int N = 5;

    if(argc > 1) {
        if(atoi(argv[1]) > 0) {
            N = atoi(argv[1]);
        } else {
            printf("\033[01;31mWARNING: You are trying to simulate a negative ");
            printf("number of wave functions.\nWill fall back to %d (default)\033[22;m\n",N);
        }
    }

    create_particle(&p, N, 0.003f);
    init_curses();
    usage(argv[0]);

    run(argc, argv);
    return 0;
}
