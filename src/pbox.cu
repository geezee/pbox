#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <math.h>
#include <time.h>

#define GL_GLEXT_PROTOTYPES
#include <GL/glut.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>


#define PI   3.141592653589793
#define L    1.0            // the length of the box
#define DIM  512            // the dimensions of the window (should be 2^n)
#define E    (L/DIM)        // epsilon
#define HBAR 1.054          // without the e-34


#define MAX(a,b) a > b ? a : b
#define MIN(a,b) a < b ? a : b


/* The struct that describes a particle */
typedef struct particle {
    float mass;            // mass of the particle
    int energy_levels;     // the number of energy levels
    float *probabilities;  // the probabilitis of every energy level
} particle;


/** global variables used in the code */
hipGraphicsResource *resource;
GLuint   buffer;
float    INCREASE_TIME = 0.01;           // number to increase the time by
particle p;                              // the particle
float    t             = INCREASE_TIME;  // starting time
int      frames        = 0;              // number of frames
float    total_time    = 0.0f;           // total time
int      PAUSE         = 0;              // whether the animation is paused


/**
 * Function headers
*/
float max(float*, int);
void  next_probabilities(float, int, float*);
void  create_particle(particle*, int, float);
float probability(particle*, float, float);
float max_probability(particle*);
void  initGL(int*, char**);
void  display();
void  key(unsigned char, int, int);
void  free_resources();
void  createVBO(GLuint*, hipGraphicsResource**, unsigned int);
void  runCuda(hipGraphicsResource**);
void  launch_kernel(uchar4);
void  runCuda(hipGraphicsResource **resource);
void  run(int, char**);
void  usage(char*);


/**
 * CUDA kernel that finds the maximum in buckets of numbers
 * and fills it in the partialMax array using the reduction method
 *
 * @param float*            the numbers to find the maximum of
 * @param int               the length of the array (param 1)
 * @param float*            list of buckets containing the maximums
*/
__global__
void cuda_max(float *numbers, int N, float *partialMax) {
    extern __shared__ float cache[];
    int cacheIndex = threadIdx.x;
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    float m = 0;
    while(tid < N) {
        m = MAX(m, numbers[tid]);
        tid += blockDim.x * gridDim.x;
    }

    cache[cacheIndex] = m;
    __syncthreads();

    int i = blockDim.x/2;
    while(i != 0) {
        if(cacheIndex < i)
            cache[cacheIndex] = MAX(cache[cacheIndex], cache[cacheIndex + i]);
        __syncthreads();

        i /= 2;
    }

    if(cacheIndex == 0)
        partialMax[blockIdx.x] = cache[0];
}

/**
 * Function that finds the maximum of an array of numbers using the
 * cuda_max function
 *
 * @param float*            the array of numbers to find the maximum of
 * @param int               the length of the array
 * @return float            the maximum of the array
*/
float max(float *numbers, int N) {
    int tpb = 256; // threads per block
    int bpg = MIN(32, (N+tpb-1)/tpb); // blocks per grid

    float *partialMax = (float *) malloc(bpg*sizeof(float));
    float *devNumbers, *devPartialMax;

    hipMalloc((void**) &devNumbers, N*sizeof(float));
    hipMalloc((void**) &devPartialMax, bpg*sizeof(float));
    
    hipMemcpy(devNumbers, numbers, N*sizeof(float), hipMemcpyHostToDevice);
    cuda_max<<<tpb, bpg, bpg*sizeof(float)>>>(devNumbers, N, devPartialMax);
    hipMemcpy(partialMax, devPartialMax, bpg*sizeof(float), hipMemcpyDeviceToHost);

    float m = partialMax[0];
    for(int i=1;i<bpg;i++)
        m = MAX(m, partialMax[i]);

    hipFree(devNumbers);
    hipFree(devPartialMax);
    free(partialMax);
    free(numbers);
    return m;
}

/**
 * The set of probabilities can be expressed as a bunch of sine waves
 * For more information read section 4.4 (Next set of probabilities) on
 * page 9 in the paper
 *
 * @param float               the time
 * @param int                 the length of the array
 * @param float*              where to store the next set of probabilities
*/
void next_probabilities(float t, int N, float* probabilities) {
    float sum = 0;
    for(int i=0;i<N;i++) {
        probabilities[i] = abs(sin(pow(10, 1-i)*t));
        sum += probabilities[i];
    }
    // Normalize the probabilities
    for(int i=0;i<N;i++) probabilities[i] /= sum;
}

/**
 * A function that creates a new particle
 *
 * @param particle*           the particle to create
 * @param int                 the number of energy levels the particle can have
 * @param float               the mass of the particle
*/
void create_particle(particle *p, int N, float mass) {
    p->mass = mass;
    p->energy_levels = N;
    p->probabilities = (float*) malloc(N*sizeof(float));
    for(int i=0;i<N;i++)
        p->probabilities[i] = (i == 0); // 1 if i == 0, 0 otherwise
                                        // so sum(probabilities) = 1
}

/**
 * CUDA device function that finds the probability of finding the
 * particle in one dimension at the position x at the energy level n
 *
 * @param int                  the energy level of the particle
 * @param float                the position of the particle
 * @return float               the probability
*/
__device__
float cuda_probability_1d_device(int n, float x) {
    float s =  E/L - (1/(n*PI)) * cos(2*n*PI*x/L) * sin(n*PI / DIM);
    return s;
}

/**
 * CUDA device function that finds the probabiltiy of finding the 
 * particle at a fixed position given a set of probabilities, the
 * number of energy levels and the position
 *
 * @param float*                the probability of each energy level
 * @param int                   the number of energy levels
 * @param float                 the x-coordinate of the particle
 * @param float                 the y-coordinate of the particle
 * @return float                the probability
*/
__device__
float cuda_probability_2d_device(float *probability, int n, float x, float y) {
    float s = 0;
    for(int i=0;i<n;i++) {
        s += (1.0/(L*L) * cuda_probability_1d_device(i+1, x) *
                          cuda_probability_1d_device(i+1, y)) * probability[i];
    }
    return s;
}

/**
 * CUDA kernel that maps the coordinate array to a probability array
 *
 * @param float*                 the probability set of each energy level
 * @param int                    the number of energy levels
 * @param float*                 the array to map to
*/
__global__ 
void cuda_probability_to_map(float *probabilities, int n, float *map) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = gridDim.x * blockDim.x;

    while(i < DIM * DIM) {
        float x = 1.0*L*((int) (i%DIM))/DIM;
        float y = 1.0*L*((int) (i/DIM))/DIM;

        if(x > 0 && x < L && y > 0 && y < L)
            map[i] = cuda_probability_2d_device(probabilities, n, x, y);
        else
            map[i] = 0;
        i += offset;
    }
}

/**
 * CUDA kernel to find the probability of finding the particle at a certain
 * position
 *
 * @param float*                  the probability set of each energy level
 * @param int                     the number of energy levels
 * @param float                   the x-coordinate of the particle
 * @param float                   the y-coordinate of the particle
 * @param float*                  used to write the probability to
*/
__global__
void cuda_probability(float *p, int N, float x, float y, float *probability) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = blockDim.x * gridDim.x;

    if(i == 0)
        probability[0] = 0;
    __syncthreads();
    
    while(i < N) {
        if(x > 0 && x < L && y > 0 && y < L) {
            atomicAdd(&probability[0],  ( 1.0/(L*L) *
                                          cuda_probability_1d_device(i+1, x) *
                                          cuda_probability_1d_device(i+1, y)
                                        ) * p[i]);
        } else {
            probability[0] = 0;
        }
        i += offset;
    }
}

/**
 * A function that finds the probability of finding the particle at a certain position
 * @param particle* p               the particle
 * @param float                     the x-coordinate of the particle
 * @param float                     the y-coordinate of the particle
 * @return                          the probability
*/
float probability(particle *p, float x, float y) {
    float *devProbabilities, *devProbability, *probability;

    hipMalloc((void**) &devProbabilities, p->energy_levels*sizeof(float));
    hipMalloc((void**) &devProbability, sizeof(float));
    probability = (float*) malloc(sizeof(float));

    hipMemcpy(devProbabilities, p->probabilities, p->energy_levels*sizeof(float), hipMemcpyHostToDevice);
    cuda_probability<<<1, p->energy_levels>>>(devProbabilities, p->energy_levels, x, y, devProbability);
    hipMemcpy(probability, devProbability, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(devProbabilities);
    hipFree(devProbability);
    float *proba;
    proba = &probability[0];
    free(probability);
    return *proba;
}

/**
 * A function that finds the maximum probability in the space of finding the particle
 * This function is used to map the probabilities [0, max] |-> [0, 255]
 * 
 * @param particle*                  the particle
 * @return                           the highest probability
*/
float max_probability(particle *p) {
    float *devProbabilities, *devMap;
    float *map;

    map = (float*) malloc(DIM * DIM * sizeof(float));
    hipMalloc((void**) &devProbabilities, p->energy_levels * sizeof(float));
    hipMalloc((void**) &devMap, DIM * DIM * sizeof(float));
    
    hipMemcpy(devProbabilities, p->probabilities, p->energy_levels * sizeof(float), hipMemcpyHostToDevice);
    cuda_probability_to_map<<<32, 256>>>(devProbabilities, p->energy_levels, devMap);
    hipMemcpy(map, devMap, DIM * DIM * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(devProbabilities);
    hipFree(devMap);

    return max(map, DIM*DIM);

    /*
    // This algorithm failed for many waves
    // I had to fall back to the original bruteforce algorithm written above
    float probability = E/L, p_1d = 0.0;
    int n;
    for(int i=0;i<10;i++) {
        n = i+1;
        p_1d = E/L + 1.0/(n*PI) * sin(n*PI*E);
        probability += p->probabilities[i] * p_1d * p_1d;
    }

    return probability;
    */
}

/**
 * A function that finds the energy of the particle at a precise energy level
 *
 * @param float                     the mass of the particle
 * @param int                       the energy level
*/
__device__
float energy(float mass, int n) {
    return (HBAR * HBAR * PI * PI * n * n) / (mass * L * L);
}

/**
 * A function that finds the highest energy the particle can reach
 * This function is useful for color mapping
 *
 * @param float                     the mass of the particle
 * @return float                    the highest energy
*/
__device__
float highest_energy(float mass, int n) {
    return energy(mass, n);
}

__global__
void kernel(uchar4 *ptr, float *probabilities, int N, float max_proba) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = blockDim.x * gridDim.x;

    while(i < DIM*DIM) {
        float x = L*((int) (i%DIM))/DIM;
        float y = L*((int) (i/DIM))/DIM;
        float p = cuda_probability_2d_device(probabilities, N, x, y)/max_proba;
        float e = 0;
        for(int j=0;j<N;j++) e += probabilities[j]*energy(1.0, j+1);
        e /= highest_energy(1.0, N);

        ptr[i].x = 255*p*e;
        ptr[i].y = 20*p;
        ptr[i].z = 255*p*(1-e);

        i += offset;
    }
}


/////////////////////////// GUI PART /////////////////////////////////////
/**
 * Initialize the OpenGL environment
 *
 * @param int       length of next paramater
 * @param char      the parameters to the environment
*/
void initGL(int *argc, char **argv) {
    glutInit(argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(DIM, DIM);
    glutCreateWindow("Particle in a box simulation");
    glutDisplayFunc(display);

    glClearColor(0.0, 0.0, 0.0, 1.0);
    glDisable(GL_DEPTH_TEST);
}

/**
 * The display function that runs on every iteration
*/
void display() {
    hipEvent_t start, stop;
    hipEventCreate(&stop);
    hipEventCreate(&start);
    hipEventRecord(start, 0);
    
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, buffer);
    glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, DIM*DIM*4, NULL, GL_DYNAMIC_DRAW_ARB);
    hipGraphicsGLRegisterBuffer(&resource, buffer, cudaGraphicsMapFlagsNone);
    glClear(GL_COLOR_BUFFER_BIT);
    runCuda(&resource);
    glDrawPixels(DIM, DIM, GL_RGBA, GL_UNSIGNED_BYTE, 0);

    glutSwapBuffers();
    t += (1-PAUSE)*INCREASE_TIME;

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float diff;
    hipEventElapsedTime(&diff, start, stop);

    glutPostRedisplay();

    total_time += diff;
    frames++;

    printf("%-5.3f (Average time per frame %.5f ms) (+%.3f)\r",
           t, total_time/frames, INCREASE_TIME);

    hipEventDestroy(start);
    hipEventDestroy(stop);

}

/**
 * Manage key strokes
 *
 * @param unsigned char         the character pressed
 * @param int                   x-location of the pressign
 * @param int                   y-location of the pressign
*/
void key(unsigned char k, int x, int y) {
    switch(k) {
        case 27:
            free_resources();
            printf("\n");
            exit(0);
        case '.':
            INCREASE_TIME *= 1.05; break;
        case ',':
            INCREASE_TIME *= 0.95; break;
        case '0':
            t = 0.01f; break;
        case ' ':
            PAUSE = 1-PAUSE; if(PAUSE == 1) glutPostRedisplay(); break;
    }
}

/**
 * Free the OpenGL resources
*/
void free_resources() {
    hipGraphicsUnregisterResource(resource);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);
    glDeleteBuffers(1, &buffer);
}

/**
 * Function that creates the buffer and the resource for the environment
 *
 * @param GLuint                    the buffer used by OpenGL
 * @param hipGraphicsResource      the cuda resource to link to the buffer
*/
void createVBO(GLuint *buffer, hipGraphicsResource **resource,
               unsigned int flags) {
    glGenBuffers(1, buffer);
    glBindBuffer(GL_ARRAY_BUFFER, *buffer);

    unsigned int size = DIM * DIM * 4 * sizeof(float);
    glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    hipGraphicsGLRegisterBuffer(resource, *buffer, flags);
}

/**
 * Kernel launcher
 *
 * @param uchar4                the pixel array
*/
void launch_kernel(uchar4 *pos) {
    float *devProbabilities;
    int N = p.energy_levels;
    hipMalloc((void**) &devProbabilities, N*sizeof(float));
    hipMemcpy(devProbabilities, p.probabilities, N*sizeof(float), hipMemcpyHostToDevice);
    kernel<<<32, 256>>>(pos, devProbabilities, N, max_probability(&p));
    next_probabilities(t, N, (p.probabilities));
    hipFree(devProbabilities);
}

/**
 * Function that creates the resources for the kernel and launches it
 *
 * @param hipGraphicsResource  the cuda resource
*/
void runCuda(hipGraphicsResource **resource) {
    uchar4 *devPtr;
    size_t size;

    hipGraphicsMapResources(1, resource, 0);
    hipGraphicsResourceGetMappedPointer((void**) &devPtr, &size, *resource);
    launch_kernel(devPtr);
    hipGraphicsUnmapResources(1, resource, 0);
}


/**
 * The function that runs everything
 *
 * @param int       length of next paramater
 * @param char      the parameters to the environment
*/
void run(int argc, char **argv) {
    hipDeviceProp_t prop;
    int dev;
    memset(&prop, 0, sizeof(hipDeviceProp_t));
    prop.major = 1;
    prop.minor = 0;
    hipChooseDevice(&dev, &prop);
    cudaGLSetGLDevice(dev);

    initGL(&argc, argv);

    glutDisplayFunc(display);
    glutKeyboardFunc(key);
    createVBO(&buffer, &resource, cudaGraphicsMapFlagsWriteDiscard);
    runCuda(&resource);

    glutMainLoop();
}



void usage(char* program_name) {
    printf("A particle in a box simulation\n");
    printf("Usage: %s n\n", program_name);
    printf("n\tThe number of energy levels to simulate (default is 5)\n\n");
    printf("Pressing these following keys will:\n");
    printf(".\tIncrease the time delay\n");
    printf(",\tDescrease the time delay\n");
    printf("<space>\tToggle pausing\n");
    printf("0\tReset the animation\n");
    printf("<esc>\tQuit\n\n");
}


///////////////////////////// MAIN FUNCTION //////////////////////////////////
int main(int argc, char *argv[]) { 
    usage(argv[0]);
    int N = 5;
    if(argc > 1) {
        if(atoi(argv[1]) > 0)
            N = atoi(argv[1]);
        else
            printf("\033[01;31mWARNING: You are trying to simulate a negative");
            printf("number of wave functions. Will fall back to %d (default)\033[22;m\n",N);
    }
    printf("\033[22;32mSimulating with %d wave functions\033[22;m\n", N);
    create_particle(&p, N, 0.003f);
    run(argc, argv);
    return 0;
}
